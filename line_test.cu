#include "hip/hip_runtime.h"
#include "include/snch_lbvh/lbvh.cuh"
#include "include/snch_lbvh/scene.cuh"
#include "include/snch_lbvh/scene_loader.cuh"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <vector>
#include <fstream>
#include <iostream>

struct PCG32
{
    uint64_t state;
    uint64_t sequence;

    __host__ __device__ PCG32(uint64_t initstate, uint64_t initseq)
    {
        state = 0U;
        sequence = (initseq << 1u) | 1u;
        (*this)();
        state += initstate;
        (*this)();
    }

    __host__ __device__ float operator()()
    {
        uint64_t oldstate = state;
        state = oldstate * 6364136223846793005ULL + sequence;
        uint32_t xorshifted = static_cast<uint32_t>(((oldstate >> 18u) ^ oldstate) >> 27u);
        uint32_t rot = static_cast<uint32_t>(oldstate >> 59u);
        uint32_t randomInt = (xorshifted >> rot) | (xorshifted << ((-rot) & 31));
        return static_cast<float>(randomInt) / static_cast<float>(UINT32_MAX);
    }
};

int main(int argc, char *argv[])
{
    if (argc < 5)
    {
        std::cerr << "Usage: " << argv[0] << " <filename> <primitive/silhouette/intersection/sample> <scene_scale> <color_scale>" << std::endl;
        return 1;
    }
    std::string filename = argv[1];
    lbvh::scene_loader<2> loader(filename);
    const auto &vertices = loader.get_vertices();
    const auto &indices = loader.get_indices();
    lbvh::scene<2> scene(vertices.begin(), vertices.end(), indices.begin(), indices.end());
    scene.compute_silhouettes();
    scene.build_bvh();
    const auto bvh_dev = scene.get_bvh_device_ptr();

    int height = 400, width = 400;
    int N = height * width;
    thrust::device_vector<float> result(N);

    std::string mode = argv[2];
    float scale = std::stof(argv[3]);
    float color_scale = std::stof(argv[4]);
    if (mode == "primitive")
    {
        thrust::transform(
            thrust::make_counting_iterator<unsigned int>(0), thrust::make_counting_iterator<unsigned int>(N),
            result.begin(),
            [bvh_dev, width, height, scale] __device__(const unsigned int idx)
            {
                float x = (static_cast<float>(idx % width) / static_cast<float>(width)) * 2.0f - 1.0f;
                float y = (static_cast<float>(idx / width) / static_cast<float>(height)) * 2.0f - 1.0f;
                float2 coord = make_float2(x * scale, y * scale);
                const auto nest = lbvh::query_device(bvh_dev, lbvh::nearest(coord), lbvh::scene<2>::distance_calculator());
                return nest.second;
            });
    }
    else if (mode == "silhouette")
    {
        thrust::transform(
            thrust::make_counting_iterator<unsigned int>(0), thrust::make_counting_iterator<unsigned int>(N),
            result.begin(),
            [bvh_dev, width, height, scale] __device__(const unsigned int idx)
            {
                float x = (static_cast<float>(idx % width) / static_cast<float>(width)) * 2.0f - 1.0f;
                float y = (static_cast<float>(idx / width) / static_cast<float>(height)) * 2.0f - 1.0f;
                float2 coord = make_float2(x * scale, y * scale);
                const auto dest = lbvh::query_device(bvh_dev, lbvh::nearest_silhouette(coord, false), lbvh::scene<2>::silhouette_distance_calculator());
                return dest;
            });
    }
    else if (mode == "intersection")
    {
        if (argc < 6)
        {
            std::cerr << "Usage: " << argv[0] << " <filename> intersection <scene_scale> <color_scale> <probe_angle>" << std::endl;
            return 1;
        }
        float angle = std::stof(argv[5]);
        thrust::transform(
            thrust::make_counting_iterator<unsigned int>(0), thrust::make_counting_iterator<unsigned int>(N),
            result.begin(),
            [bvh_dev, width, height, scale, angle] __device__(const unsigned int idx)
            {
                float x = (static_cast<float>(idx % width) / static_cast<float>(width)) * 2.0f - 1.0f;
                float y = (static_cast<float>(idx / width) / static_cast<float>(height)) * 2.0f - 1.0f;
                float2 coord = make_float2(x * scale, y * scale);
                // auto li = lbvh::line_intersect(lbvh::Line<float, 2>(coord, lbvh::normalize(make_float2(1.0f, 1.0f))));
                const auto dest = lbvh::query_device(
                    bvh_dev,
                    lbvh::line_intersect(lbvh::Line<float, 2>(coord, lbvh::normalize(make_float2(std::cos(angle / 180.0f * M_PI), std::sin(angle / 180.0f * M_PI))))),
                    lbvh::scene<2>::intersect_test());
                if (dest.first == false)
                {
                    return 1.0f;
                }
                else
                {
                    return dest.second;
                }
            });
    }
    else if (mode == "sample")
    {
        if (argc < 6)
        {
            std::cerr << "Usage: " << argv[0] << " <filename> intersection <scene_scale> <color_scale> <radius>" << std::endl;
            return 1;
        }
        float radius = std::stof(argv[5]);
        thrust::transform(
            thrust::make_counting_iterator<unsigned int>(0), thrust::make_counting_iterator<unsigned int>(N),
            result.begin(),
            [bvh_dev, width, height, scale, radius] __device__(unsigned int idx)
            {
                PCG32 pcg(42, 64 * idx);
                float x = (static_cast<float>(idx % width) / static_cast<float>(width)) * 2.0f - 1.0f;
                float y = (static_cast<float>(idx / width) / static_cast<float>(height)) * 2.0f - 1.0f;
                float2 coord = make_float2(x * scale, y * scale);
                float ret = 0.0f;
                float u = pcg();
                const auto sample_result = lbvh::sample_object_in_sphere(
                    bvh_dev,
                    lbvh::sphere_intersect(lbvh::sphere<float, 2>(coord, radius)),
                    lbvh::scene<2>::intersect_sphere(),
                    lbvh::scene<2>::measurement_getter(),
                    lbvh::scene<2>::green_weight(),
                    u);
                const int object_idx = sample_result.first;
                if (object_idx == -1)
                {
                    return 1.0f;
                }
                float v = pcg();
                const float2 sample_point = lbvh::sample_on_object(
                    bvh_dev,
                    object_idx,
                    lbvh::scene<2>::sample_on_object(),
                    u, v);
                ret += lbvh::length(make_float2(coord.x - sample_point.x, coord.y - sample_point.y));
                return ret;
            });
    }
    else
    {
        std::cout << "Invalid mode." << std::endl;
        exit(1);
    }

    thrust::host_vector<float> host_result = result;
    std::ofstream file("output.ppm");
    file << "P3\n"
         << width << " " << height << "\n255\n";
    for (int i = 0; i < N; i++)
    {
        int gray_value = static_cast<int>(host_result[i] / color_scale * 255);
        gray_value = std::max(0, std::min(255, gray_value));
        file << gray_value << " " << gray_value << " " << gray_value << "\n";
    }
    file.close();
    std::cout << "Image savd to output.ppm" << std::endl;

    return 0;
}